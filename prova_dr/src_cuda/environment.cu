#include "hip/hip_runtime.h"
#include "environment.cuh"
#include "utils.h"

void Environment::generateSinusoidalSurface(float picks_depth, int density){

  // generate a "super dense" cloud of points expressed in camera_r frame
  float left_bound=-picks_depth/3-(0.1*picks_depth);
  float right_bound=(picks_depth/3)+(0.1*picks_depth);

  for (int x=0; x<density; x++)
    for (int y=0; y<density; y++){
      float x_ = ((float)x/(float)density)*(right_bound-left_bound)+left_bound;
      float y_ = ((float)y/(float)density)*(-left_bound-left_bound)+left_bound;

      float depth = ((sin((x_)*(6*3.14))*sin((x_)*(6*3.14))+sin((y_)*(6*3.14))*sin((y_)*(6*3.14)))/2.0);

      // int clr_x = ((float)x/(float)density)*255*(sin((x_)*(6*3.14))*sin((x_)*(6*3.14)));
      // int clr_y = ((float)y/(float)density)*255*(sin((y_)*(6*3.14))*sin((y_)*(6*3.14)));
      // int clr_z = depth*(255.0/picks_depth);

      unsigned char clr_x = ((float)x/(float)density)*255*depth;
      unsigned char clr_y = ((float)y/(float)density)*255*depth;
      unsigned char clr_z = depth*(255.0/picks_depth);


      Cp cp;
      cp.point=Eigen::Vector3f(x_,y_,depth);
      cp.color[0]=clr_x;
      cp.color[1]=clr_y;
      cp.color[2]=clr_z;
      cp_vector_.push_back(cp);
    }

  cp_d_ = Environment::getCpPtrOnGPU();

}

Cp* Environment::getCpPtrOnGPU(){

  hipError_t err ;

  Cp* cp_vector_h = &cp_vector_[0];
  Cp* cp_vector_d;

  hipMalloc(&cp_vector_d, sizeof(Cp)*cp_vector_.size());
  err = hipGetLastError();
  if (err != hipSuccess)
      printf("hipMalloc cp Error: %s\n", hipGetErrorString(err));

  hipMemcpy(cp_vector_d, cp_vector_h, sizeof(Cp)*cp_vector_.size(), hipMemcpyHostToDevice);
  err = hipGetLastError();
  if (err != hipSuccess)
      printf("hipMemcpy cp Error: %s\n", hipGetErrorString(err));

  return cp_vector_d;
}


void Environment::generateCamera(std::string name, float t1, float t2, float t3, float alpha1, float alpha2, float alpha3){
  Eigen::Vector3f t_r(t1,t2,t3);
  Eigen::Isometry3f* frame_world_wrt_camera_r = new Eigen::Isometry3f;
  frame_world_wrt_camera_r->linear().setIdentity();  //TODO implement orientation
  frame_world_wrt_camera_r->translation()=t_r;
  Eigen::Isometry3f* frame_camera_wrt_world_r = new Eigen::Isometry3f;
  *frame_camera_wrt_world_r = frame_world_wrt_camera_r->inverse();
  Camera_cpu* camera = new Camera_cpu(name,lens_,aspect_,film_,resolution_,max_depth_,frame_camera_wrt_world_r,frame_world_wrt_camera_r);
  camera_vector_cpu_.push_back(camera);
  Camera_gpu* camera_d = camera->getCamera_gpu();
  camera_vector_gpu_.push_back(camera_d);

}
