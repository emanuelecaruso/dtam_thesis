#include "camera_cpu.cuh"
#include <thread>
#include <vector>
#include <mutex>

using namespace std;
using namespace pr;



void Camera_cpu::gpuFree(){
  image_rgb_gpu_.release();
  depth_map_gpu_.release();
}


Camera_gpu* Camera_cpu::getCamera_gpu(){

  image_rgb_gpu_.upload(image_rgb_->image_);
  depth_map_gpu_.upload(depth_map_->image_);

  Camera_gpu* camera_gpu_h = new Camera_gpu(name_, lens_, aspect_, width_, resolution_,
     max_depth_, K_, Kinv_, *frame_camera_wrt_world_, *frame_world_wrt_camera_,
      depth_map_gpu_, image_rgb_gpu_);

  hipError_t err ;

  Camera_gpu* camera_gpu_d;
  hipMalloc((void**)&camera_gpu_d, sizeof(Camera_gpu));
  err = hipGetLastError();
  if (err != hipSuccess)
      printf("hipMalloc %s%s",name_," Error: %s\n", hipGetErrorString(err));

  hipMemcpy(camera_gpu_d, camera_gpu_h, sizeof(Camera_gpu), hipMemcpyHostToDevice);
  err = hipGetLastError();
  if (err != hipSuccess)
      printf("hipMemcpy %s%s",name_," Error: %s\n", hipGetErrorString(err));

  delete camera_gpu_h;

  return camera_gpu_d;
}

void Camera_cpu::cloneCameraImages(Camera* camera){
  depth_map_ = camera->depth_map_;
  image_rgb_ = camera->image_rgb_;

}

void Camera_cpu::showInvdepthmap(int scale){
  Image< float >* invdepthmap = new Image< float >;
  camera_r->depth_map_gpu_.download(invdepthmap->image_);
  invdepthmap->image_=1.0/(2.0*(invdepthmap->image_));
  invdepthmap->show(scale/resolution_);
}
