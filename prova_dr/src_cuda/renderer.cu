#include "hip/hip_runtime.h"
#include "renderer.cuh"
#include <thread>
#include <vector>
#include <mutex>

bool Renderer::renderPoint(Cp& cp, Camera_cpu* camera){

  Eigen::Vector2f uv;
  float depth_cam;
  bool point_in_front_of_camera = camera->projectPoint(cp.point, uv, depth_cam );
  if (!point_in_front_of_camera)
    return false;

  float width = camera->width_;
  float height = camera->width_/camera->aspect_;

  if(uv.x()<0 || uv.x()>width)
    return false;
  if(uv.y()<0 || uv.y()>height)
    return false;

  Eigen::Vector2i pixel_coords;
  camera->uv2pixelCoords( uv, pixel_coords);

  float depth = depth_cam/camera->max_depth_;

  float evaluated_pixel;
  camera->depth_map_->evalPixel(pixel_coords,evaluated_pixel);

  if (evaluated_pixel<depth)
    return false;

  if (depth>1 || depth>255 || cp.color[0]>255 || cp.color[1]>255 || cp.color[2]>255)
    return false;

  cv::Vec3b color = cv::Vec3b(cp.color[0],cp.color[1],cp.color[2]);

  camera->image_rgb_->setPixel(pixel_coords, color);
  camera->depth_map_->setPixel(pixel_coords,depth);

  return true;
}

__global__ void renderPoint_gpu(Cp* cp, Camera_gpu* camera_gpu_d ){
// __global__ void renderPoint_gpu( ){

  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  bool valid = true;
  Eigen::Vector2f uv; float depth_cam;

  bool point_in_front_of_camera = camera_gpu_d->projectPoint(cp[i].point, uv, depth_cam );
  if (!point_in_front_of_camera)
    valid = false;

  float width = camera_gpu_d->width_;
  float height = camera_gpu_d->width_/camera_gpu_d->aspect_;

  if(uv.x()<0 || uv.x()>width)
    valid = false;
  if(uv.y()<0 || uv.y()>height)
    valid = false;

  Eigen::Vector2i pixel_coords;
  camera_gpu_d->uv2pixelCoords( uv, pixel_coords);

  float depth = depth_cam/camera_gpu_d->max_depth_;

  // printf("\n");
  // printf("%f",cp[i].point.x());
  // printf("%f",cp[i].point.y());
  // printf("%f",cp[i].point.z());
  // printf("\n");

  float evaluated_pixel = camera_gpu_d->depth_map_(pixel_coords.y(),pixel_coords.x());

  if (evaluated_pixel<depth)
    valid = false;

  if (depth>1 || depth>255 || cp[i].color[0]>255 || cp[i].color[1]>255 || cp[i].color[2]>255)
    valid = false;

  if (valid){
    uchar3 color = make_uchar3( cp[i].color[0], cp[i].color[1], cp[i].color[2] );
    camera_gpu_d->image_rgb_(pixel_coords.y(),pixel_coords.x())= color;
    camera_gpu_d->depth_map_(pixel_coords.y(),pixel_coords.x())= depth;
  }
}

void Renderer::renderImage_naive(cpVector& cp_vector, Camera_cpu* camera){

    camera->clearImgs();
    for (Cp cp : cp_vector)
    {
      Renderer::renderPoint(cp, camera);
    }

}
bool Renderer::renderImages_parallel_gpu(Environment* environment){

  hipError_t err ;
  Cp* cp_d = environment->cp_d_;
  int cp_size = environment->cp_vector_.size();

  int numThreads = 32;
  int numBlocks = cp_size / numThreads;

  if (cp_size % numThreads != 0)
    return false;

  for (int i=0; i<environment->camera_vector_cpu_.size(); i++){
    Camera_cpu* camera_cpu = environment->camera_vector_cpu_[i];
    Camera_gpu* camera_gpu = environment->camera_vector_gpu_[i];


    renderPoint_gpu<<<numBlocks,numThreads>>>( cp_d, camera_gpu );
    // renderPoint_gpu<<<1,1>>>( cp_d, camera_gpu_d );
    err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error executing rendering kernel: %s\n", hipGetErrorString(err));

    // auto a = camera_cpu->image_rgb_gpu_;

    camera_cpu->image_rgb_gpu_.download(camera_cpu->image_rgb_->image_);
    camera_cpu->depth_map_gpu_.download(camera_cpu->depth_map_->image_);
    // camera_cpu->image_rgb_gpu_.download(image_rgb_->image_);
    // hipMemcpy(valid_h, valid_d, sizeof(bool), hipMemcpyDeviceToHost);
    // std::cout << *valid_h << std::endl;

  }



  return true;

}
